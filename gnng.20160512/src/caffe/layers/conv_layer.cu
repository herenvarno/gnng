#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_layer.hpp"

namespace caffe {



template <typename Dtype>
void ConvolutionLayer<Dtype>::LayerSetUp(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  // Configure the kernel size, padding, stride, and inputs.
  ConvolutionParameter conv_param = this->layer_param_.convolution_param();
  force_nd_im2col_ = conv_param.force_nd_im2col();
  channel_axis_ = bottom[0]->CanonicalAxisIndex(conv_param.axis());
  const int first_spatial_axis = channel_axis_ + 1;
  const int num_axes = bottom[0]->num_axes();
  num_spatial_axes_ = num_axes - first_spatial_axis;
  CHECK_GE(num_spatial_axes_, 0);
  vector<int> bottom_dim_blob_shape(1, num_spatial_axes_ + 1);
  vector<int> spatial_dim_blob_shape(1, std::max(num_spatial_axes_, 1));
  // Setup filter kernel dimensions (kernel_shape_).
  kernel_shape_.Reshape(spatial_dim_blob_shape);
  int* kernel_shape_data = kernel_shape_.mutable_cpu_data();
  if (conv_param.has_kernel_h() || conv_param.has_kernel_w()) {
    CHECK_EQ(num_spatial_axes_, 2)
        << "kernel_h & kernel_w can only be used for 2D convolution.";
    CHECK_EQ(0, conv_param.kernel_size_size())
        << "Either kernel_size or kernel_h/w should be specified; not both.";
    kernel_shape_data[0] = conv_param.kernel_h();
    kernel_shape_data[1] = conv_param.kernel_w();
  } else {
    const int num_kernel_dims = conv_param.kernel_size_size();
    CHECK(num_kernel_dims == 1 || num_kernel_dims == num_spatial_axes_)
        << "kernel_size must be specified once, or once per spatial dimension "
        << "(kernel_size specified " << num_kernel_dims << " times; "
        << num_spatial_axes_ << " spatial dims).";
      for (int i = 0; i < num_spatial_axes_; ++i) {
        kernel_shape_data[i] =
            conv_param.kernel_size((num_kernel_dims == 1) ? 0 : i);
      }
  }
  for (int i = 0; i < num_spatial_axes_; ++i) {
    CHECK_GT(kernel_shape_data[i], 0) << "Filter dimensions must be nonzero.";
  }
  // Setup stride dimensions (stride_).
  stride_.Reshape(spatial_dim_blob_shape);
  int* stride_data = stride_.mutable_cpu_data();
  if (conv_param.has_stride_h() || conv_param.has_stride_w()) {
    CHECK_EQ(num_spatial_axes_, 2)
        << "stride_h & stride_w can only be used for 2D convolution.";
    CHECK_EQ(0, conv_param.stride_size())
        << "Either stride or stride_h/w should be specified; not both.";
    stride_data[0] = conv_param.stride_h();
    stride_data[1] = conv_param.stride_w();
  } else {
    const int num_stride_dims = conv_param.stride_size();
    CHECK(num_stride_dims == 0 || num_stride_dims == 1 ||
          num_stride_dims == num_spatial_axes_)
        << "stride must be specified once, or once per spatial dimension "
        << "(stride specified " << num_stride_dims << " times; "
        << num_spatial_axes_ << " spatial dims).";
    const int kDefaultStride = 1;
    for (int i = 0; i < num_spatial_axes_; ++i) {
      stride_data[i] = (num_stride_dims == 0) ? kDefaultStride :
          conv_param.stride((num_stride_dims == 1) ? 0 : i);
      CHECK_GT(stride_data[i], 0) << "Stride dimensions must be nonzero.";
    }
  }
  // Setup pad dimensions (pad_).
  pad_.Reshape(spatial_dim_blob_shape);
  int* pad_data = pad_.mutable_cpu_data();
  if (conv_param.has_pad_h() || conv_param.has_pad_w()) {
    CHECK_EQ(num_spatial_axes_, 2)
        << "pad_h & pad_w can only be used for 2D convolution.";
    CHECK_EQ(0, conv_param.pad_size())
        << "Either pad or pad_h/w should be specified; not both.";
    pad_data[0] = conv_param.pad_h();
    pad_data[1] = conv_param.pad_w();
  } else {
    const int num_pad_dims = conv_param.pad_size();
    CHECK(num_pad_dims == 0 || num_pad_dims == 1 ||
          num_pad_dims == num_spatial_axes_)
        << "pad must be specified once, or once per spatial dimension "
        << "(pad specified " << num_pad_dims << " times; "
        << num_spatial_axes_ << " spatial dims).";
    const int kDefaultPad = 0;
    for (int i = 0; i < num_spatial_axes_; ++i) {
      pad_data[i] = (num_pad_dims == 0) ? kDefaultPad :
          conv_param.pad((num_pad_dims == 1) ? 0 : i);
    }
  }
  // Setup dilation dimensions (dilation_).
  dilation_.Reshape(spatial_dim_blob_shape);
  int* dilation_data = dilation_.mutable_cpu_data();
  const int num_dilation_dims = conv_param.dilation_size();
  CHECK(num_dilation_dims == 0 || num_dilation_dims == 1 ||
        num_dilation_dims == num_spatial_axes_)
      << "dilation must be specified once, or once per spatial dimension "
      << "(dilation specified " << num_dilation_dims << " times; "
      << num_spatial_axes_ << " spatial dims).";
  const int kDefaultDilation = 1;
  for (int i = 0; i < num_spatial_axes_; ++i) {
    dilation_data[i] = (num_dilation_dims == 0) ? kDefaultDilation :
                       conv_param.dilation((num_dilation_dims == 1) ? 0 : i);
  }
  // Special case: im2col is the identity for 1x1 convolution with stride 1
  // and no padding, so flag for skipping the buffer and transformation.
  is_1x1_ = true;
  for (int i = 0; i < num_spatial_axes_; ++i) {
    is_1x1_ &=
        kernel_shape_data[i] == 1 && stride_data[i] == 1 && pad_data[i] == 0;
    if (!is_1x1_) { break; }
  }
  // Configure output channels and groups.
  channels_ = bottom[0]->shape(channel_axis_);
  num_output_ = this->layer_param_.convolution_param().num_output();
  CHECK_GT(num_output_, 0);
  group_ = this->layer_param_.convolution_param().group();
  CHECK_EQ(channels_ % group_, 0);
  CHECK_EQ(num_output_ % group_, 0)
      << "Number of output should be multiples of group.";
  if (reverse_dimensions()) {
    conv_out_channels_ = channels_;
    conv_in_channels_ = num_output_;
  } else {
    conv_out_channels_ = num_output_;
    conv_in_channels_ = channels_;
  }
  // Handle the parameters: weights and biases.
  // - blobs_[0] holds the filter weights
  // - blobs_[1] holds the biases (optional)
  vector<int> weight_shape(2);
  weight_shape[0] = conv_out_channels_;
  weight_shape[1] = conv_in_channels_ / group_;
  for (int i = 0; i < num_spatial_axes_; ++i) {
    weight_shape.push_back(kernel_shape_data[i]);
  }
  bias_term_ = this->layer_param_.convolution_param().bias_term();
  vector<int> bias_shape(bias_term_, num_output_);
  if (this->blobs_.size() > 0) {
    CHECK_EQ(1 + bias_term_, this->blobs_.size())
        << "Incorrect number of weight blobs.";
    if (weight_shape != this->blobs_[0]->shape()) {
      Blob<Dtype> weight_shaped_blob(weight_shape);
      LOG(FATAL) << "Incorrect weight shape: expected shape "
          << weight_shaped_blob.shape_string() << "; instead, shape was "
          << this->blobs_[0]->shape_string();
    }
    if (bias_term_ && bias_shape != this->blobs_[1]->shape()) {
      Blob<Dtype> bias_shaped_blob(bias_shape);
      LOG(FATAL) << "Incorrect bias shape: expected shape "
          << bias_shaped_blob.shape_string() << "; instead, shape was "
          << this->blobs_[1]->shape_string();
    }
    LOG(INFO) << "Skipping parameter initialization";
  } else {
    if (bias_term_) {
      this->blobs_.resize(2);
    } else {
      this->blobs_.resize(1);
    }
    // Initialize and fill the weights:
    // output channels x input channels per-group x kernel height x kernel width
    this->blobs_[0].reset(new Blob<Dtype>(weight_shape));
    shared_ptr<Filler<Dtype> > weight_filler(GetFiller<Dtype>(
        this->layer_param_.convolution_param().weight_filler()));
    weight_filler->Fill(this->blobs_[0].get());
    // If necessary, initialize and fill the biases.
    if (bias_term_) {
      this->blobs_[1].reset(new Blob<Dtype>(bias_shape));
      shared_ptr<Filler<Dtype> > bias_filler(GetFiller<Dtype>(
          this->layer_param_.convolution_param().bias_filler()));
      bias_filler->Fill(this->blobs_[1].get());
    }
  }
  kernel_dim_ = this->blobs_[0]->count(1);
  weight_offset_ = conv_out_channels_ * kernel_dim_ / group_;
  // Propagate gradients to the parameters (as directed by backward pass).
  this->param_propagate_down_.resize(this->blobs_.size(), true);
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Reshape(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const int first_spatial_axis = channel_axis_ + 1;
  CHECK_EQ(bottom[0]->num_axes(), first_spatial_axis + num_spatial_axes_)
      << "bottom num_axes may not change.";
  num_ = bottom[0]->count(0, channel_axis_);
  CHECK_EQ(bottom[0]->shape(channel_axis_), channels_)
      << "Input size incompatible with convolution kernel.";
  // TODO: generalize to handle inputs of different shapes.
  for (int bottom_id = 1; bottom_id < bottom.size(); ++bottom_id) {
    CHECK(bottom[0]->shape() == bottom[bottom_id]->shape())
        << "All inputs must have the same shape.";
  }
  // Shape the tops.
  bottom_shape_ = &bottom[0]->shape();
  compute_output_shape();
  vector<int> top_shape(bottom[0]->shape().begin(),
      bottom[0]->shape().begin() + channel_axis_);
  top_shape.push_back(num_output_);
  for (int i = 0; i < num_spatial_axes_; ++i) {
    top_shape.push_back(output_shape_[i]);
  }
  for (int top_id = 0; top_id < top.size(); ++top_id) {
    top[top_id]->Reshape(top_shape);
  }
  if (reverse_dimensions()) {
    conv_out_spatial_dim_ = bottom[0]->count(first_spatial_axis);
  } else {
    conv_out_spatial_dim_ = top[0]->count(first_spatial_axis);
  }
  col_offset_ = kernel_dim_ * conv_out_spatial_dim_;
  output_offset_ = conv_out_channels_ * conv_out_spatial_dim_ / group_;
  // Setup input dimensions (conv_input_shape_).
  vector<int> bottom_dim_blob_shape(1, num_spatial_axes_ + 1);
  conv_input_shape_.Reshape(bottom_dim_blob_shape);
  int* conv_input_shape_data = conv_input_shape_.mutable_cpu_data();
  for (int i = 0; i < num_spatial_axes_ + 1; ++i) {
    if (reverse_dimensions()) {
      conv_input_shape_data[i] = top[0]->shape(channel_axis_ + i);
    } else {
      conv_input_shape_data[i] = bottom[0]->shape(channel_axis_ + i);
    }
  }
  // The im2col result buffer will only hold one image at a time to avoid
  // overly large memory usage. In the special case of 1x1 convolution
  // it goes lazily unused to save memory.
  col_buffer_shape_.clear();
  col_buffer_shape_.push_back(kernel_dim_ * group_);
  for (int i = 0; i < num_spatial_axes_; ++i) {
    if (reverse_dimensions()) {
      col_buffer_shape_.push_back(input_shape(i + 1));
    } else {
      col_buffer_shape_.push_back(output_shape_[i]);
    }
  }
  col_buffer_.Reshape(col_buffer_shape_);
  bottom_dim_ = bottom[0]->count(channel_axis_);
  top_dim_ = top[0]->count(channel_axis_);
  num_kernels_im2col_ = conv_in_channels_ * conv_out_spatial_dim_;
  num_kernels_col2im_ = reverse_dimensions() ? top_dim_ : bottom_dim_;
  // Set up the all ones "bias multiplier" for adding biases by BLAS
  out_spatial_dim_ = top[0]->count(first_spatial_axis);
  if (bias_term_) {
    vector<int> bias_multiplier_shape(1, out_spatial_dim_);
    bias_multiplier_.Reshape(bias_multiplier_shape);
    caffe_set(bias_multiplier_.count(), Dtype(1),
        bias_multiplier_.mutable_cpu_data());
  }
}


template <typename Dtype>
void ConvolutionLayer<Dtype>::forward_gpu_gemm(const Dtype* input,
    const Dtype* weights, Dtype* output, bool skip_im2col) {
  const Dtype* col_buff = input;
  if (!is_1x1_) {
    if (!skip_im2col) {
      conv_im2col_gpu(input, col_buffer_.mutable_gpu_data());
    }
    col_buff = col_buffer_.gpu_data();
  }
  
  for (int g = 0; g < group_; ++g) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, conv_out_channels_ /
        group_, conv_out_spatial_dim_, kernel_dim_,
        (Dtype)1., weights + weight_offset_ * g, col_buff + col_offset_ * g,
        (Dtype)0., output + output_offset_ * g);
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::forward_gpu_bias(Dtype* output,
    const Dtype* bias) {
  caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num_output_,
      out_spatial_dim_, 1, (Dtype)1., bias, bias_multiplier_.gpu_data(),
      (Dtype)1., output);
}


/*
 * THE KERNEL TO DO ALL THE CONV OPERATION
 */
template <typename Dtype>
__global__ void conv_do_gpu_kernel(const int n,
		const Dtype* I, const int I_cs, const int I_h, const int I_w,
    	const int kernel_h, const int kernel_w,
    	const int pad_h, const int pad_w,
    	const int stride_h, const int stride_w,
    	const int dilation_h, const int dilation_w,
    	const Dtype* W, const int W_h, const int W_w,
    	const Dtype* B, const int B_h, const int B_w,
    	Dtype* O, const int O_h, const int O_w,
    	Dtype* col_buffer) {
	CUDA_KERNEL_LOOP(index, n) {
		Dtype out=0;
		const int width_col = (I_w + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
	  	const int O_x = (index/W_h) / O_w;
	  	const int O_y = (index/W_h) % O_w;
	  	const int W_y = O_y;
	  	const int kernel_dim = kernel_w*kernel_h;
//		for(int i=0; i<W_h; i++){
//		for(int i=0; i<1; i++){
		const int i = index % W_h;
		const int W_x = i;
		const int I_x = ((O_x/width_col)-1)*stride_h + (dilation_h * (kernel_h - 1 ) + 1) - 2*pad_h + (i%kernel_dim)/kernel_w - kernel_h +1;
		const int I_y = ((O_x%width_col)-1)*stride_w + (dilation_w * (kernel_w - 1 ) + 1) - 2*pad_w + (i%kernel_dim)%kernel_w - kernel_w +1;
		//const int I_x = 1;
		//const int I_y = 1;
		const int I_c = i / kernel_dim;
			if(I_x<I_h && I_y<I_w && I_c<I_cs){
				//out = out + 1;
				out = (I[I_c*+I_h*I_w+I_x*I_w+I_y] * W[W_y*W_h+W_x]);
			}
//		}
		if(B && i==0){
			out = out + B[O_y];
		}
//		if(index==0){
		O[O_y*O_h+O_x]+=out;
//	}
	}
}

/* JUST A BACKUP
template <typename Dtype>
__global__ void conv_do_gpu_kernel(const int n,
		const Dtype* I, const int I_cs, const int I_h, const int I_w,
    	const int kernel_h, const int kernel_w,
    	const int pad_h, const int pad_w,
    	const int stride_h, const int stride_w,
    	const int dilation_h, const int dilation_w,
    	const Dtype* W, const int W_h, const int W_w,
    	const Dtype* B, const int B_h, const int B_w,
    	Dtype* O, const int O_h, const int O_w,
    	Dtype* col_buffer) {
	CUDA_KERNEL_LOOP(index, n) {
		Dtype out=0;
		
		const int width_col = (I_w + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
	
		const int idx = index / W_h;
	  	const int O_x = idx / O_w;
	  	const int O_y = idx % O_w;
	  	const int W_y = O_y;
	  	const int T_x = O_x;
			int T_y = index % W_h;
			int W_x = index % W_h;
			int I_x = ((T_x/width_col)-1)*stride_h + (dilation_h * (kernel_h - 1 ) + 1) - 2*pad_h + (T_y%(kernel_w*kernel_h))/kernel_w - kernel_h +1;
			int I_y = ((T_x%width_col)-1)*stride_w + (dilation_w * (kernel_w - 1 ) + 1) - 2*pad_w + (T_y%(kernel_w*kernel_h))%kernel_w - kernel_w +1;
	  		int I_c = T_y / (kernel_h * kernel_w);
			if(I_x<I_h && I_y<I_w && I_c<I_cs){
				out = (I[I_c*+I_h*I_w+I_x*I_w+I_y] * W[W_y*W_h+W_x]);
			}
		if(B && index<B_w){
			out = out + B[index];
		}
//		O[O_y*O_h+O_x]=out;
	}
}
*/

template <typename Dtype>
void ConvolutionLayer<Dtype>::forward_gpu_do(const Dtype* input,
    const Dtype* weights, const Dtype* bias, Dtype* output) {
  if (!is_1x1_) {
//		int height = conv_input_shape_.cpu_data()[1];
//		int width = conv_input_shape_.cpu_data()[2];
//		int kernel_h = kernel_shape_.cpu_data()[0];
//		int kernel_w = kernel_shape_.cpu_data()[1];
//		int pad_h = pad_.cpu_data()[0];
//		int pad_w = pad_.cpu_data()[1];
//		int stride_h = stride_.cpu_data()[0];
//		int stride_w = stride_.cpu_data()[1];
//		int dilation_h = dilation_.cpu_data()[0];
//		int dilation_w = dilation_.cpu_data()[1];
//		int num_output = num_output_;
		int channels = conv_in_channels_ * kernel_shape_.cpu_data()[0] * kernel_shape_.cpu_data()[1] ;
	  // We are going to launch channels * height_col * width_col kernels, each
	  // kernel responsible for copying a single-channel grid.
		int height_col = (conv_input_shape_.cpu_data()[1] + 2 * pad_.cpu_data()[0] -
		  (dilation_.cpu_data()[0] * (kernel_shape_.cpu_data()[0] - 1) + 1)) / stride_.cpu_data()[0] + 1;
		int width_col = (conv_input_shape_.cpu_data()[2] + 2 * pad_.cpu_data()[1] -
		  (dilation_.cpu_data()[1] * (kernel_shape_.cpu_data()[1] - 1) + 1)) / stride_.cpu_data()[1] + 1;
		int num_kernels = channels* height_col * width_col * num_output_;
		
		
	  // NOLINT_NEXT_LINE(whitespace/operators)
	  conv_do_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
	                         CAFFE_CUDA_NUM_THREADS>>>(
		  num_kernels, input, conv_in_channels_, conv_input_shape_.cpu_data()[1], conv_input_shape_.cpu_data()[2], kernel_shape_.cpu_data()[0], kernel_shape_.cpu_data()[1], pad_.cpu_data()[0],
		  pad_.cpu_data()[1], stride_.cpu_data()[0], stride_.cpu_data()[1], dilation_.cpu_data()[0], dilation_.cpu_data()[1], weights, channels, num_output_, bias, height_col*width_col, num_output_, output, height_col * width_col, num_output_, col_buffer_.mutable_gpu_data());
	  CUDA_POST_KERNEL_CHECK;
  }
}



template <typename Dtype>
void ConvolutionLayer<Dtype>::compute_output_shape() {
  const int* kernel_shape_data = this->kernel_shape_.cpu_data();
  const int* stride_data = this->stride_.cpu_data();
  const int* pad_data = this->pad_.cpu_data();
  const int* dilation_data = this->dilation_.cpu_data();
  this->output_shape_.clear();
  for (int i = 0; i < this->num_spatial_axes_; ++i) {
    // i + 1 to skip channel axis
    const int input_dim = this->input_shape(i + 1);
    const int kernel_extent = dilation_data[i] * (kernel_shape_data[i] - 1) + 1;
    const int output_dim = (input_dim + 2 * pad_data[i] - kernel_extent)
        / stride_data[i] + 1;
    this->output_shape_.push_back(output_dim);
  }
}

template <typename Dtype>
void ConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* weight = this->blobs_[0]->gpu_data();

  for (int i = 0; i < bottom.size(); ++i) {	
     const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    
   for (int n = 0; n < this->num_; ++n) {	// NOTE: For each one in mini batch
   
   /*
    * HERE IS CAFFE'S CODE
    */
 /*        this->forward_gpu_gemm(bottom_data + n * this->bottom_dim_, weight,
          top_data + n * this->top_dim_);
      if (this->bias_term_) {
        const Dtype* bias = this->blobs_[1]->gpu_data();
        this->forward_gpu_bias(top_data + n * this->top_dim_, bias);
      }*/
    /*
     * HERE IS MY CODE
     */
    if(this->bias_term_){
     	const Dtype* bias = this->blobs_[1]->gpu_data();
      	this->forward_gpu_do(bottom_data + n * this->bottom_dim_, weight, bias, top_data + n * this->top_dim_);
      }else{
      	this->forward_gpu_do(bottom_data + n * this->bottom_dim_, weight, NULL, top_data + n * this->top_dim_);
      }
      hipDeviceSynchronize();
    }
  }
}

INSTANTIATE_CLASS(ConvolutionLayer);
REGISTER_LAYER_CLASS(Convolution);

}  // namespace caffe
